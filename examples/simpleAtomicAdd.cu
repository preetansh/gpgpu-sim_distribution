#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void
baseline_atomic_add_kernel(int* finished) {
	int i = 0;
	while (i < 10) {
		// use atomic CAS to set finished to 0
        atomicAdd(finished, 1);
        i++;
	}
}


int main(int argc, char *argv[])
{
  	printf("Simple Atomic Add kernel\n");

  	int* finished;
	finished = (int *) malloc(sizeof(int));
	(*finished) = 1;
	int* device_finished;
	hipMalloc(&device_finished, 1 * sizeof(int));
	hipMemcpy(device_finished, finished, 1 * sizeof(int), hipMemcpyHostToDevice);

	// compute number of blocks and threads per block
    const int threadsPerBlock = 1;
    const int blocks = 1;

	baseline_atomic_add_kernel<<<blocks, threadsPerBlock>>>(device_finished);

	hipDeviceSynchronize();
    hipMemcpy(finished, device_finished, 1 * sizeof(int), hipMemcpyDeviceToHost);

    printf("Final finished %d\n", *finished);
}