#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void
baseline_atomic_add_kernel(int* finished) {
	int i = 0;
	while (i < 10) {
		// use atomic CAS to set finished to 0
        atomicAdd(finished, 1);
        i++;
	}
}

__global__ void
baseline_atomic_cas_kernel(int* finished) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	printf("Spinning before : %d\n", index);
	while (atomicCAS(finished, 0, 1) != 0) {
		// printf("Spinning inside : %d\n", index);
	}
	// printf("Spinning outside : %d\n", index);
    atomicExch(finished, 0);
}

__global__ void
baseline_atomic_cas_2_kernel(int* finished, int* total) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	bool done = false;
	printf("Spinning before : %d\n", index);
	while (!done) {
		printf("Spinning inside : %d\n", index);
		if (atomicCAS(finished, 0, 1) == 0) {
	    	__threadfence();
			printf("Spinning acq : %d\n", index);
	    	done = true;
	    	*total += 1;
	    	__threadfence();
	    	atomicExch(finished, 0);
		}
	}
	printf("Spinning finished : %d\n", index);
}


int main(int argc, char *argv[])
{
  	printf("Simple Atomic Add kernel\n");

  	int* finished;
  	int* total;
	finished = (int *) malloc(sizeof(int));
	(*finished) = 0;
	total = (int *) malloc(sizeof(int));
	(*total) = 0;
	int* device_finished;
	int* device_total;
	hipMalloc(&device_finished, 1 * sizeof(int));
	hipMalloc(&device_total, 1 * sizeof(int));
	hipMemcpy(device_finished, finished, 1 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_total, total, 1 * sizeof(int), hipMemcpyHostToDevice);

	// compute number of blocks and threads per block
    const int threadsPerBlock = 64;
    const int blocks = 1;

	baseline_atomic_cas_2_kernel<<<blocks, threadsPerBlock>>>(device_finished, device_total);

	hipDeviceSynchronize();
    hipMemcpy(finished, device_finished, 1 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(total, device_total, 1 * sizeof(int), hipMemcpyDeviceToHost);

    printf("Final finished %d %d\n", *finished, *total);
}